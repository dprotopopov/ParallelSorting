﻿char *title = "bucket sort";
char *description = "Блочная сортировка (bucket sort)";
/*
В блочной карманной или корзинной сортировке(Bucket sort) сортируемые элементы распределены
между конечным числом отдельных блоков(карманов, корзин).Каждый блок затем сортируется отдельно либо
рекурсивно тем же методом либо другим. Затем элементы помещают обратно в массив.
Для этой сортировки характерно линейное время исполнения.
Алгоритм требует знаний о природе сортируемых данных, выходящих за рамки функций "сравнить" и
"поменять местами", достаточных для сортировки слиянием, сортировки пирамидой, быстрой сортировки,
сортировки Шелла, сортировки вставкой.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#define assert( bool ) 

template<class T> __device__ void device_exchange(T *x, T *y, int count);
template<class T>__device__ void device_copy(T *x, T *y, int count);
template<class T> __device__ int device_comparer(T *x, T *y);
template<class T> __device__ int device_indexator(T *x, int index, int len);
template<class T> __device__ void device_bubble_sort(T *data, int index, int len, int n, int direction);
template<class T> __global__ void global_bucket_worker_collect(T * data, T * bucket, int * sizes, int n, int index,	int len, int direction);
template<class T> __global__ void global_bucket_worker_sort(T * data, T * bucket, int * sizes, int n, int index,	int len, int direction);
template<class T> __global__ void global_bucket_worker_merge(T * data, T * bucket, int * sizes, int n, int index,	int len, int direction);
template<class T> __host__ void host_bucket_sort(T *data, int n, int direction);

////////////////////////////////////////////////////////////////////////////////////////////
// Настроечные аттрибуты
// _comparer - функция сравнения двух элементов массива
// _indexator - функция определения номера корзины для элемента массива
// _non_parallel_sort - фунция сортировки без использования паралельных вычислений
// _parallel_sort - фунция сортировки с использованием паралельных вычислений

#define fn_comparer  device_comparer<long>
#define fn_indexator device_indexator<long>
#define fn_non_parallel_sort device_bubble_sort<long>
#define fn_parallel_sort host_bucket_sort<long>

template<class T>
__host__ void host_bucket_sort(T *data, int n, int direction)
{
// data - массив данных
// n - количество элементов в исходном массиве для сортировки
// direction - способ сортировки 
// -1 означает сортировку по убыванию, 
//  1 означает сортировку по возрастанию
	
	hipError_t err;
	T *device_data;
	T *device_bucket;
	int *device_size;

	// Шаг первый - копируем исходный массив в память GPU 

	err = hipMalloc((void**)&device_data, n*sizeof(T));
	hipMemcpy(device_data, data, n*sizeof(T), hipMemcpyHostToDevice);

	// Определим оптимальноe количество корзин и парамерты индексатора

	int number = 2; while (number<n && number<5) number++;
	
	int len = 1 ; while ((1<<len)<number) len++;
	int index = 8*sizeof(T)-len;

	// Шаг второй - выделяем память под корзины 
	
	err = hipMalloc((void**)&device_bucket, (n<<len)*sizeof(T));
	err = hipMalloc((void**)&device_size, (1<<len)*sizeof(int));

	// Определим оптимальное разбиения на процессы, нити

	int blocks = 1 << (len>>1);
	int threads = 1 << (len-(len>>1));

	// Шаг трeтий - применяем алгоритм

	assert((1<<len) == blocks*threads);

	global_bucket_worker_collect <<< blocks, threads >>>(device_data, device_bucket, device_size, n, index, len, direction);
	global_bucket_worker_sort <<< blocks, threads >>>(device_data, device_bucket, device_size, n, index, len, direction);
	global_bucket_worker_merge <<< 1, 1 >>>(device_data, device_bucket, device_size, n, index, len, direction);

	// Возвращаем результаты в исходный массив

	hipMemcpy(data, device_data, n*sizeof(T), hipMemcpyDeviceToHost);

	// Освобождаем память на устройстве

	hipFree(device_data);
	hipFree(device_bucket);
	hipFree(device_size);

	err = err;
}

// Функция процесса
// Параметры
//	адрес массива данных 
//	адрес массива корзин 
//	адрес массива размера корзин 
//	Размер массива
//  Параметры индексатора
//  Размер одного элемента
//	Направление сортировки
template<class T>
__global__ void global_bucket_worker_collect(
	T * data, 
	T * bucket,
	int * sizes,
	int n,
	int index,
	int len,
	int direction)
{
	// Получаем идентификатор нити
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	if (id < (1<<len)) {
		sizes[id] = 0;
		// Набираем товары в корзину
		for(int i = 0; i < n; i++) {
			if (id==fn_indexator(&data[i],index,len)) {
				device_copy(&bucket[id*n+sizes[id]++],&data[i],1);
			}
		}
	}
}
template<class T>
__global__ void global_bucket_worker_sort(
	T * data, 
	T * bucket,
	int * sizes,
	int n,
	int index,
	int len,
	int direction)
{
	// Получаем идентификатор нити
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	if (id < (1<<len)) {
		fn_non_parallel_sort(bucket, id*n, sizes[id], n, direction);
	}
}
template<class T>
__global__ void global_bucket_worker_merge(
	T * data, 
	T * bucket,
	int * sizes,
	int n,
	int index,
	int len,
	int direction)
{
	// Получаем идентификатор нити
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	assert(id == 0);
	id = id;

	T * next = data;
	for(int i=0; i < (1 << len) ; i++ ) {
		device_copy(next,&bucket[i*n],sizes[i]);
		next = &next[sizes[i]];
	}
}

// Перестановка двух блоков в памяти устройства
template<class T>
__device__ void device_exchange(T *x, T *y, int count)
{
	for(int i = 0; i < count ; i++ ) {
		T ch = x[i] ; x[i] = y[i] ; y[i] = ch;
	}
}

// Копирование одного участка памяти в другой
template<class T>
__device__ void device_copy(T *x, T *y, int count)
{
	for(int i = 0; i < count ; i++ ) {
		x[i] = y[i] ;
	}
}

// Определение номера карзины 
// Формируется положмтельное число из len бит с позиции index
template<class T>
__device__ int device_indexator(T *x, int index, int len)
{
	assert(index+len <= sizeof(T));
	return (int)((((*x) >> index) + (1 << (8 * sizeof(T)-index))) & ((1 << len) - 1));
}

// Функция сравнения данных xранимых в памяти как целых чисел типа long
// comparison function which returns ​a negative integer value if the first argument is less than the second, 
// a positive integer value if the first argument is greater than the second and zero if the arguments are equal.
template<class T>
__device__ int device_comparer(T *x, T *y)
{
	if ((*x)<(*y)) return -1;
	else if ((*x)>(*y)) return 1;
	else return 0;
}

/////////////////////////////////////////////////////////////////
// Пузырьковая сортировка части массива
// Особенность - поддерживат циклическую адресацию в массиве длины n
template<class T>
__device__ void device_bubble_sort(T *data, int index, int len, int n, int direction)
{
	if (index+len <= n) {
		for(int i = index ; i < index+len-1 ; i++ ) {
			for(int j = i + 1 ; j < index+len ; j++ ) {
				int value = direction*fn_comparer(&data[i],&data[j]);
				if (value > 0) device_exchange<T>(&data[i],&data[j],1);
			}
		}
	} else {
		for(int i = 0 ; i < ((index+len) % n) ; i++ ) {
			for(int j = i + 1 ; j <= ((index+len)%n) ; j++ ) {
				int value = direction*fn_comparer(&data[i],&data[j]);
				if (value > 0) device_exchange<T>(&data[i],&data[j],1);
			}
			for(int j = index ; j < n ; j++ ) {
				int value = direction*fn_comparer(&data[i],&data[j]);
				if (value > 0) device_exchange<T>(&data[i],&data[j],1);
			}
		}
		for(int i = index ; i < n-1 ; i++ ) {
			for(int j = i + 1 ; j < n ; j++ ) {
				int value = direction*fn_comparer(&data[i],&data[j]);
				if (value > 0) device_exchange<T>(&data[i],&data[j],1);
			}
		}
	}
}

int main(int argc, char* argv[])
{
	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	for (int n = 100, tests = 100; n <= 1000; n += 100, tests = ((tests>>1)+10))
	{
		// Создаём массив длины n чисел типа long
		long *arr = (long *)malloc(n*sizeof(long));

		float total_time = 0.0;
		bool check = true;

		for(int j = 0; j < tests ; j++ ) {
			// Заполняем массив псевдо-случайными значениями используя функцию rand
			for (int i = 0; i<n; i++) { arr[i] = rand(); }

			// Сортируем массив по возрастанию
		
			time_t start = time(NULL);
			fn_parallel_sort(arr, n, 1);
			time_t end = time(NULL);

			total_time += (end - start);

			// Проверяем
			for (int i = 0; (i < (n - 1)) && check; i++)
				check = (arr[i] <= arr[i + 1]);
		}
		std::cout << "array size = " << n << "\t" << "avg time = " << (total_time/tests) << "\t" << "check result = " << (check ? "ok" : "fail") << "\t";
		for (int i = 0; i<n && i<24; i++) std::cout << arr[i] << ","; std::cout << " ..." << std::endl;

		// Высвобождаем массив
		free(arr);
	}

	hipDeviceReset();

	exit(0);
}